
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#define MAXN 20000

int* get_data(char* file_name){
	FILE *f = fopen(file_name,"r");
	int *a = (int *)malloc(MAXN*sizeof(int));
	for(int i=0; i<MAXN; i++){
		fscanf(f,"%d",&a[i]);
	}
	return a;
}

int get_sum(int *a, int n){

	int sum=0;
	for(int i=0; i<n; i++)
		sum+=a[i];

	return sum;
}


__global__ void reduce(int *g_idata, int *res, int num_elems){

	int tid = threadIdx.x;
	
	int n = num_elems/blockDim.x;


	// after this step we have reduced 100000 to 1000 elems and we have 1K total threads
	res[tid]=0;
	for(int i=0; i<n; i++)
		res[tid] += g_idata[tid*n+i];
	__syncthreads();


	// start with N/2 threads and keep movivng up the tree
	// finally result is obtained in res[0];
	for(int step=1; step<blockDim.x; step<<=1)
	{
		int idx = 2*tid*step;

		if(idx+step < blockDim.x){

			res[idx] += res[idx+step];
		}
		__syncthreads();
	}

}


int main(){
	int *h_data, *d_data, *res;
	int result;
	int num_elems = MAXN;
	int size = num_elems*sizeof(int);
	h_data = get_data((char *)"reduce_data.txt");
	int threads_per_block = 1000;
	int num_blocks = 1;
	dim3 dimBlock(threads_per_block);
	dim3 dimGrid(num_blocks);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipMalloc(&d_data, size);
	hipMalloc(&res, threads_per_block*sizeof(int));
	hipMemcpy(d_data, h_data, size, hipMemcpyHostToDevice);

	hipEventRecord(start);
	reduce<<<dimGrid, dimBlock>>>(d_data, res, num_elems);
	hipEventRecord(stop);
	hipMemcpy(&result, res, sizeof(int), hipMemcpyDeviceToHost);
	hipEventSynchronize(stop);
	float exec_time = 0;
	hipEventElapsedTime(&exec_time, start, stop);
	
	printf("Cuda Reduction result: %d\n",result);
	int sum = get_sum(h_data, num_elems);
	printf("Actual answer: %d\n",sum);
	printf("Kernel execution time in msec: %f\n", exec_time);
	hipFree(d_data);
	free(h_data);
}
